#include "hip/hip_runtime.h"
//
// Created by kouushou on 2021/6/6.
//
#include "Mytime.h"
#include <cstdio>

const int blockSIZE = 512;

__global__ void reductionNeighbored(double *idata, double *outdata, unsigned int n) {
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    double *curData = idata + blockIdx.x * blockDim.x;
    if (idx >= n)return;

    for (unsigned stride = 1; stride < blockDim.x; stride <<= 1u) {

        if (!(tid % (stride << 1u))) {
            curData[tid] += curData[stride + tid];
        }
        __syncthreads();
    }
    if (tid == 0) {
        outdata[blockIdx.x] = curData[0];
    }
}

double calReductionGPU_Neighbored(const double *a, unsigned siz, double *time) {
    double *d_val;
    double *d_val_OUT;
    double *h_val_OUT;
    unsigned int nnz = 1u << siz;
    dim3 block(blockSIZE, 1);
    dim3 grid((nnz + blockSIZE - 1) / blockSIZE, 1);

    h_val_OUT = (double *) malloc(sizeof(double) * grid.x);

    hipMalloc(&d_val_OUT, sizeof(double) * grid.x);
    hipMalloc(&d_val, sizeof(double) * nnz);

    hipMemcpy(d_val, a, sizeof(double) * nnz, hipMemcpyHostToDevice);


    MyTimeStart();
    reductionNeighbored<<<grid, block>>>(d_val, d_val_OUT, nnz);

    hipDeviceSynchronize();
    *time = MyTimePassed();

    hipMemcpy(h_val_OUT, d_val_OUT, sizeof(double) * grid.x, hipMemcpyDeviceToHost);
    double res = 0;
    for (int i = 0; i < grid.x; ++i) {
        res += h_val_OUT[i];
    }
    free(h_val_OUT);
    hipFree(d_val_OUT);
    hipFree(d_val);
    return res;
}

__global__ void reductionNeighboredNoDivided(double *idata, double *outdata, unsigned int n) {
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    double *curData = idata + blockIdx.x * blockDim.x;
    if (idx >= n)return;

    for (unsigned stride = 1; stride < blockDim.x; stride <<= 1u) {

        if (!(tid & ((stride << 1u) - 1u))) {
            curData[tid] += curData[stride + tid];
        }
        __syncthreads();
    }
    if (tid == 0) {
        outdata[blockIdx.x] = curData[0];
    }
}

double calReductionGPU_NeighboredNoDivided(const double *a, unsigned siz, double *time) {
    double *d_val;
    double *d_val_OUT;
    double *h_val_OUT;
    unsigned int nnz = 1u << siz;
    dim3 block(blockSIZE, 1);
    dim3 grid((nnz + blockSIZE - 1) / blockSIZE, 1);

    h_val_OUT = (double *) malloc(sizeof(double) * grid.x);

    hipMalloc(&d_val_OUT, sizeof(double) * grid.x);
    hipMalloc(&d_val, sizeof(double) * nnz);

    hipMemcpy(d_val, a, sizeof(double) * nnz, hipMemcpyHostToDevice);


    MyTimeStart();
    reductionNeighboredNoDivided<<<grid, block>>>(d_val, d_val_OUT, nnz);

    hipDeviceSynchronize();
    *time = MyTimePassed();

    hipMemcpy(h_val_OUT, d_val_OUT, sizeof(double) * grid.x, hipMemcpyDeviceToHost);
    double res = 0;
    for (int i = 0; i < grid.x; ++i) {
        res += h_val_OUT[i];
    }
    free(h_val_OUT);
    hipFree(d_val_OUT);
    hipFree(d_val);
    return res;
}


__global__ void reductionNeighboredLessToRight(double *idata, double *outdata, unsigned int n) {
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    double *curData = idata + blockIdx.x * blockDim.x;
    if (idx >= n)return;

    for (unsigned stride = 1; stride < blockDim.x; stride <<= 1u) {


        unsigned index = 2u * stride * tid;
        if (index < blockDim.x) {
            curData[index] += curData[stride + index];
        }

        __syncthreads();
    }
    if (tid == 0) {
        outdata[blockIdx.x] = curData[0];
    }
}

double calReductionGPU_NeighboredLessToRight(const double *a, unsigned siz, double *time) {
    double *d_val;
    double *d_val_OUT;
    double *h_val_OUT;
    unsigned int nnz = 1u << siz;
    dim3 block(blockSIZE, 1);
    dim3 grid((nnz + blockSIZE - 1) / blockSIZE, 1);

    h_val_OUT = (double *) malloc(sizeof(double) * grid.x);

    hipMalloc(&d_val_OUT, sizeof(double) * grid.x);
    hipMalloc(&d_val, sizeof(double) * nnz);

    hipMemcpy(d_val, a, sizeof(double) * nnz, hipMemcpyHostToDevice);


    MyTimeStart();
    reductionNeighboredLessToRight<<<grid, block>>>(d_val, d_val_OUT, nnz);

    hipDeviceSynchronize();
    *time = MyTimePassed();

    hipMemcpy(h_val_OUT, d_val_OUT, sizeof(double) * grid.x, hipMemcpyDeviceToHost);
    double res = 0;
    for (int i = 0; i < grid.x; ++i) {
        res += h_val_OUT[i];
    }
    free(h_val_OUT);
    hipFree(d_val_OUT);
    hipFree(d_val);
    return res;
}

__global__ void reductionNeighboredReverse(double *idata, double *outdata, unsigned int n) {
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    double *curData = idata + blockIdx.x * blockDim.x;
    if (idx >= n)return;

    for (unsigned stride = blockDim.x>>1u; stride; stride >>= 1u) {

        if(tid < stride){
            curData[tid]+=curData[tid+stride];
        }

        __syncthreads();
    }
    if (tid == 0) {
        outdata[blockIdx.x] = curData[0];
    }
}

double calReductionGPU_NeighboredReverse(const double *a, unsigned siz, double *time) {
    double *d_val;
    double *d_val_OUT;
    double *h_val_OUT;
    unsigned int nnz = 1u << siz;
    dim3 block(blockSIZE, 1);
    dim3 grid((nnz + blockSIZE - 1) / blockSIZE, 1);

    h_val_OUT = (double *) malloc(sizeof(double) * grid.x);

    hipMalloc(&d_val_OUT, sizeof(double) * grid.x);
    hipMalloc(&d_val, sizeof(double) * nnz);

    hipMemcpy(d_val, a, sizeof(double) * nnz, hipMemcpyHostToDevice);


    MyTimeStart();
    reductionNeighboredReverse<<<grid, block>>>(d_val, d_val_OUT, nnz);

    hipDeviceSynchronize();
    *time = MyTimePassed();

    hipMemcpy(h_val_OUT, d_val_OUT, sizeof(double) * grid.x, hipMemcpyDeviceToHost);
    double res = 0;
    for (int i = 0; i < grid.x; ++i) {
        res += h_val_OUT[i];
    }
    free(h_val_OUT);
    hipFree(d_val_OUT);
    hipFree(d_val);
    return res;
}

